// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "cuda_sgemm_cublas.cuh"
#include "cuda_sgemm_v10.cuh"

#define M 4096
#define K 4096
#define N 4096


int main()
{
	hstAllocMatrix(h_a, M, K);
	hstAllocMatrix(h_b, K, N);

	hstInitMatrix(h_a);
	hstInitMatrix(h_b);

	hstEchoMatrix(h_a, 4);
	hstEchoMatrix(h_b, 4);

	kerAllocMatrix(d_a, M, K);
	kerAllocMatrix(d_b, K, N);
	kerAllocMatrix(d_c, M, N);

    checkCudaErrors(hipMemcpy(d_a.data, h_a.data, M * K * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b.data, h_b.data, K * N * sizeof(float), hipMemcpyHostToDevice));

    // Calculate with cublas
    hstAllocMatrix(h_ref, M, N);
    hstCalByRef(d_a, d_b, d_c);
    checkCudaErrors(hipMemcpy(h_ref.data, d_c.data, M * N * sizeof(float), hipMemcpyDeviceToHost));
    hstEchoMatrix(h_ref, 4);

    // Calculate with cuda_sgemm_v1
    hstAllocMatrix(h_v10, M, N);
    hstCalByV10<32>(d_a, d_b, d_c);
    checkCudaErrors(hipMemcpy(h_v10.data, d_c.data, d_c.rows * d_c.cols * sizeof(float), hipMemcpyDeviceToHost));
    hstEchoMatrix(h_v10, 4);
/*


	//



    // Calculate with shared memory blocking
*/
}



